﻿#include <hip/hip_runtime.h>

#include <stdio.h>
#include <cmath>
#include <vector>


struct Vector3 {
    float X, Y, Z;
};

__device__ Vector3 Hit(Vector3 center, int radius, Vector3 origin, Vector3 direction, float zfar)
{
    Vector3 oc = { center.X - origin.X, center.Y - origin.Y, center.Z - origin.Z };
    float a = direction.X * direction.X + direction.Y * direction.Y + direction.Z * direction.Z;
    float half_b = direction.X * oc.X + direction.Y * oc.Y + direction.Z * oc.Z;
    float c = oc.X * oc.X + oc.Y * oc.Y + oc.Z * oc.Z - radius * radius;
    float discriminant = half_b * half_b - a * c;

    if (discriminant < 0)
    {
        return { 0, 0, zfar };
    }
    float sqrtd = sqrt(discriminant);

    float root = (half_b - sqrtd) / a;
    float z1 = origin.Z + root * direction.Z;
    float y1 = origin.Y + root * direction.Y;
    float x1 = origin.X + root * direction.X;

    root = (half_b + sqrtd) / a;
    float z2 = origin.Z + root * direction.Z;
    float y2 = origin.Y + root * direction.Y;
    float x2 = origin.X + root * direction.X;

    if (z1 >= z2) return { x1, y1, z1 };
    else return { x2, y2, z2 };
}

extern "C" __global__ void RenderSpheres(float* thing, float* depth, unsigned char* canvas, int things, Vector3 camera, int radius, int cx, int cy, float zfar, Vector3 light1, Vector3 light2)
{
    float x, y, z, r, g, b;
    float xminP, yminP, xmaxP, ymaxP;
    int index;
    float shadow;
    float distance;
    float norX, norY, norZ;
    float ligX, ligY, ligZ;

    int chunksize = cx / blockDim.x;
    int xmin = chunksize * threadIdx.x;
    int xmax = chunksize * (threadIdx.x + 1);

    chunksize = cy / blockDim.y;
    int ymin = chunksize * threadIdx.y;
    int ymax = chunksize * (threadIdx.y + 1);

    for (int i = 0; i < things; i++)
    {
        x = thing[6 * i];
        y = thing[6 * i + 1];
        z = thing[6 * i + 2];
        r = thing[6 * i + 3];
        g = thing[6 * i + 4];
        b = thing[6 * i + 5];

        xminP = camera.X + (x + radius - camera.X) * (0 - camera.Z) / (z - camera.Z);
        yminP = camera.Y + (y + radius - camera.Y) * (0 - camera.Z) / (z - camera.Z);
        xmaxP = camera.X + (x - radius - camera.X) * (0 - camera.Z) / (z - camera.Z);
        ymaxP = camera.Y + (y - radius - camera.Y) * (0 - camera.Z) / (z - camera.Z);
        if (xminP < xmin || xmaxP >= xmax || yminP < ymin || ymaxP >= ymax) continue;
        
        for (int yP = ymin; yP < ymax; yP++)
        {
            for (int xP = xmin; xP < xmax; xP++)
            {
                index = xP + yP * cx;

                Vector3 center = { x, y, z };
                Vector3 direction = { xP - camera.X, yP - camera.Y, 0 - camera.Z };
                Vector3 ray = Hit(center, radius, camera, direction, zfar);
                if (ray.Z > depth[3 * index])
                {
                    depth[3 * index] = ray.Z;

                    shadow = 0;
                    norX = ray.X - x;
                    norY = ray.Y - y;
                    norZ = ray.Z - z;
                    distance = sqrt(norX * norX + norY * norY + norZ * norZ);
                    norX /= distance;
                    norY /= distance;
                    norZ /= distance;
                    ligX = xP - light1.X;
                    ligY = yP - light1.Y;
                    ligZ = 0 - light1.Z;
                    distance = sqrt(ligX * ligX + ligY * ligY + ligZ * ligZ);
                    ligX /= distance;
                    ligY /= distance;
                    ligZ /= distance;
                    shadow += -24.0f * (1 + norX * ligX + norY * ligY + norZ * ligZ);
                    ligX = xP - light2.X;
                    ligY = yP - light2.Y;
                    ligZ = 0 - light2.Z;
                    distance = sqrt(ligX * ligX + ligY * ligY + ligZ * ligZ);
                    ligX /= distance;
                    ligY /= distance;
                    ligZ /= distance;
                    shadow += -24.0f * (1 + norX * ligX + norY * ligY + norZ * ligZ);

                    canvas[4 * index] = (unsigned char)(b + shadow);
                    canvas[4 * index + 1] = (unsigned char)(g + shadow);
                    canvas[4 * index + 2] = (unsigned char)(r + shadow);
                    canvas[4 * index + 3] = 255;
                }
            }
        }
    }
}

extern "C" __global__ void RenderLight(float* thing, float* depth, unsigned char* canvas, int things, int channel, Vector3 light, int radius, int cx, int cy, float zfar)
{
    float x, y, z;
    float xminP, yminP, xmaxP, ymaxP;
    int index;

    int chunksize = cx / blockDim.x;
    int xmin = chunksize * threadIdx.x;
    int xmax = chunksize * (threadIdx.x + 1);

    chunksize = cy / blockDim.y;
    int ymin = chunksize * threadIdx.y;
    int ymax = chunksize * (threadIdx.y + 1);

    for (int i = 0; i < things; i++)
    {
        x = thing[6 * i];
        y = thing[6 * i + 1];
        z = thing[6 * i + 2];

        xminP = light.X + (x + radius - light.X) * (0 - light.Z) / (z - light.Z);
        yminP = light.Y + (y + radius - light.Y) * (0 - light.Z) / (z - light.Z);
        xmaxP = light.X + (x - radius - light.X) * (0 - light.Z) / (z - light.Z);
        ymaxP = light.Y + (y - radius - light.Y) * (0 - light.Z) / (z - light.Z);
        if (xminP < xmin || xmaxP >= xmax || yminP < ymin || ymaxP >= ymax) continue;

        for (int yP = ymin; yP < ymax; yP++)
        {
            for (int xP = xmin; xP < xmax; xP++)
            {
                index = xP + yP * cx;

                Vector3 center = { x, y, z };
                Vector3 direction = { xP - light.X, yP - light.Y, 0 - light.Z };
                Vector3 ray = Hit(center, radius, light, direction, zfar);
                if (ray.Z > depth[3 * index + channel])
                {
                    depth[3 * index + channel] = ray.Z;
                }
            }
        }
    }
}
